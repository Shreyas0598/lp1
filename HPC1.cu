#include<iostream>
#include<cstdio>
#include<cstdlib>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void Min(float* InputArray, int ArraySize){
	int tid = threadIdx.x;
	int ThreadCount = blockDim.x;

	do{
		if(tid<ThreadCount){
			if((tid + ThreadCount)<ArraySize)
				if(InputArray[tid] > InputArray[tid + ThreadCount])
					InputArray[tid] = InputArray[tid + ThreadCount];
		}

		ThreadCount = (ThreadCount+1)>>1;
		ArraySize = (ArraySize + 1)>>1;
	}while(ThreadCount>1);
	
	if(InputArray[0] > InputArray[1])
		InputArray[0] = InputArray[1];
}

__global__ void Max(float* InputArray, int ArraySize){
	int tid = threadIdx.x;
	int ThreadCount = blockDim.x;

	do{
		if(tid<ThreadCount){
			if((tid + ThreadCount)<ArraySize)
				if(InputArray[tid] < InputArray[tid + ThreadCount])
					InputArray[tid] = InputArray[tid + ThreadCount];
		}

		ThreadCount = (ThreadCount+1)>>1;
		ArraySize = (ArraySize + 1)>>1;
	}while(ThreadCount>1);
	
	if(InputArray[0] < InputArray[1])
		InputArray[0] = InputArray[1];
}

__global__ void Sum(float* InputArray, int ArraySize){
	int tid = threadIdx.x;
	int ThreadCount = blockDim.x;

	do{
		if(tid<ThreadCount){
			if((tid + ThreadCount)<ArraySize)
				InputArray[tid] += InputArray[tid + ThreadCount];
		}

		ThreadCount = (ThreadCount+1)>>1;
		ArraySize = (ArraySize + 1)>>1;
	}while(ThreadCount>1);
	
	InputArray[0] += InputArray[1];
}

__global__ void Average(float* InputArray, int ArraySize){
	int tid = threadIdx.x;
	int ThreadCount = blockDim.x;
	int TempArraySize = ArraySize;

	do{
		if(tid<ThreadCount){
			if((tid + ThreadCount)<TempArraySize)
				InputArray[tid] += InputArray[tid + ThreadCount];
		}

		ThreadCount = (ThreadCount+1)>>1;
		TempArraySize = (TempArraySize + 1)>>1;
	}while(ThreadCount>1);
	
	InputArray[0] += InputArray[1];
	
	InputArray[0] /= ArraySize;
}

int main(){
	//Read Array Size From User
	int ArraySize = -1;
	printf("Enter The Number Of Elements: : ");
	scanf("%d", &ArraySize);
	
	if(ArraySize<=0)
		return 0;
	
	//Declare The Float Array
	float *h_Array=new float[ArraySize];

	printf("Enter The Elements In The Array: : ");

	//Read Elements From User
	for(int i=0;i<ArraySize;i++){
		scanf("%f", &h_Array[i]);
	}
		
	int ArrayMemory=ArraySize*sizeof(float);
	int ThreadBlockSize = (ArraySize+1)>>1;
	
	float *d_Array;
	float result;
	
	hipMalloc(&d_Array,ArrayMemory);
	
	// Copy Array To GPU For Minimum Function
	hipMemcpy(d_Array, h_Array, ArrayMemory, hipMemcpyHostToDevice);
  	Min<<<1,ThreadBlockSize>>>(d_Array,ArraySize);
	hipMemcpy(&result, d_Array, sizeof(float), hipMemcpyDeviceToHost);
	printf("The Minimum Value In The Array: : %f\n", result);
	
	// Copy Array To GPU For Maximum Function
	hipMemcpy(d_Array, h_Array, ArrayMemory, hipMemcpyHostToDevice);
  	Max<<<1,ThreadBlockSize>>>(d_Array,ArraySize);
	hipMemcpy(&result, d_Array, sizeof(float), hipMemcpyDeviceToHost);
	printf("The Maximum Value In The Array: : %f\n", result);
	
	// Copy Array To GPU For Sum Function
	hipMemcpy(d_Array, h_Array, ArrayMemory, hipMemcpyHostToDevice);
  	Sum<<<1,ThreadBlockSize>>>(d_Array,ArraySize);
	hipMemcpy(&result, d_Array, sizeof(float), hipMemcpyDeviceToHost);
	printf("The Sum Of Numbers In The Array: : %f\n", result);
	
	// Copy Array To GPU For Average Function
	hipMemcpy(d_Array, h_Array, ArrayMemory, hipMemcpyHostToDevice);
  	Average<<<1,ThreadBlockSize>>>(d_Array,ArraySize);
	hipMemcpy(&result, d_Array, sizeof(float), hipMemcpyDeviceToHost);
	printf("The Average Of Numbers In The Array: : %f\n", result);
	
	return 0;
}
